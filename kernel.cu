#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <sstream>
#include <fstream>
#include <stdio.h>
#include <map>
#include "vector"
#include <algorithm>
#include <chrono>
using namespace std;
// Maybe implement later
//__global__ void sentimentAnalysis(string* positiveVocab, string* negativeVocab) {
//	int i = threadIdx.x;
//	
//	float positiveAssoc = 0;
//	float negativeAssoc = 0;
//	for (vector<string>::iterator it = positiveVocab.begin(); it != positiveVocab.end(); ++it) {
//		//Positive
//		countPositive();
//	
//	}
//	for (vector<string>::iterator it = negativeVocab.begin(); it != negativeVocab.end(); ++it) {
//		//Negative
//		countNegative();
//
//	}
//	semanticOrientation[item.first] = positiveAssoc - negativeAssoc;
//
//
//}

//__global__ void countPositive(string* positiveVocab) {
//	int i = threadIdx.x;
//	/*	if (*it == item.first) {
//			negativeAssoc += pmi[item.first].begin()->second;
//		}*/
//
//}
//
//__global__ void countNegative(string* negativeVocab) {
//	int i = threadIdx.x;
//	//if (*it == item.first) {
//	//	negativeAssoc += pmi[item.first].begin()->second;
//	//}
//}

__global__ void addPositive(string* positiveVocab) {
	int i = threadIdx.x;
	// count positive
	// count negative

}


//hipError_t sentimentAnalysis(vector<string> positiveVocab, vector<string> negativeVocab, vector<string> allTerms, unsigned int posSize, unsigned int negSize, unsigned int termSize) {
//	hipError_t cudaStatus;
//
//
//	string* d_positiveVocab = 0;
//	string* d_negativeVocab = 0;
//	string* d_allTerms = 0;
//
//
//	// Select GPU in PCIE slot 1 to run on.
//	cudaStatus = hipSetDevice(0);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
//		goto Error;
//	}
//
//	// Allocate GPU buffers for three vectors (two input, one output)    .
//	cudaStatus = hipMalloc((void**)&d_positiveVocab, posSize * sizeof(string));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&d_negativeVocab, negSize * sizeof(string));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMalloc((void**)&d_allTerms, termSize * sizeof(string));
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMalloc failed!");
//		goto Error;
//	}
//
//	// Copy input vectors from host memory to GPU buffers.
//	cudaStatus = hipMemcpy(d_positiveVocab, &positiveVocab, posSize * sizeof(string), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	cudaStatus = hipMemcpy(d_negativeVocab, &negativeVocab, negSize * sizeof(string), hipMemcpyHostToDevice);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//	// Launch a kernel on the GPU with one thread for each element.
//	addPositive << <1, posSize >> > (d_positiveVocab);
//
//	// Check for any errors launching the kernel
//	cudaStatus = hipGetLastError();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
//		goto Error;
//	}
//
//	// hipDeviceSynchronize waits for the kernel to finish, and returns
//	// any errors encountered during the launch.
//	cudaStatus = hipDeviceSynchronize();
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
//		goto Error;
//	}
//
//	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(d_allTerms, &allTerms, termSize * sizeof(int), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) {
//		fprintf(stderr, "hipMemcpy failed!");
//		goto Error;
//	}
//
//Error:
//	hipFree(d_positiveVocab);
//	hipFree(d_negativeVocab);
//	hipFree(d_allTerms);
//
//
//	return cudaStatus;
//}



__global__ void denomKernel(float* d_p_t_Vector, float* d_denom_Vector, int n) {
	int i = threadIdx.x;

	if (i < n) {
		d_denom_Vector[i] = d_p_t_Vector[i];
	}



}

// Helper function for using CUDA to add vectors in parallel.
hipError_t denomCuda(float* p_t_Vector, float* denom_Vector, unsigned int p_t_size, unsigned int denom_size)
{
	float* d_p_t_Vector = 0;
	float* d_denom_Vector = 0;
	hipError_t cudaStatus;
	vector<float> tempting;
	for (int i = 0; i < p_t_size; i++) {
		tempting.push_back(p_t_Vector[i]);

	}

	for (auto item : tempting) {
		cout << item << endl;
	}

	cout << tempting.data() << endl;
	cout << denom_Vector << endl;

	//cout << denom_Vector[155] << endl;
	//cout << &p_t_Vector[2] << endl;
	//cout << &denom_Vector[155] << endl;

	cout << p_t_Vector[0] << endl;



	// Choose which GPU to run on
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	cudaStatus = hipMallocManaged(&d_p_t_Vector, p_t_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_p_t_Vector, &p_t_Vector, p_t_size * sizeof(float), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}


	cudaStatus = hipMallocManaged((float**)&d_denom_Vector, denom_size * sizeof(float));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}



	// Launch a kernel on the GPU with one thread for each element.
	denomKernel <<<1, p_t_size >>> (p_t_Vector, d_denom_Vector, p_t_size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching denomKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(denom_Vector, d_denom_Vector, denom_size * sizeof(float), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(d_p_t_Vector);
	hipFree(d_denom_Vector);

	return cudaStatus;
}





// Comparator function to sort pairs
// according to second value
bool cmp(pair<string, float>& a,
	pair<string, float>& b)
{
	return a.second < b.second;
}

// Function to sort the map according
// to value in a (key-value) pairs
void sort(map<string, float>& M)
{

	// Declare vector of pairs
	vector<pair<string, float> > A;

	// Copy key-value pair from Map
	// to vector of pairs
	for (auto& it : M) {
		A.push_back(it);
	}

	sort(A.begin(), A.end(), cmp);

}



int main()
{
	string lineInput;
	string delimiter = " ";
	size_t pos = 0;
	string token;

	map <string, map<string, float>> all_com;
	map <string, float> countAll;

	vector<float> occurrenceCount;
	map <string, float> p_t;
	map <string, map<string, float>> p_t_com;
	map <string, map<string, float>> pmi;
	vector<pair<string, float>> semanticOrientation;
	int count = 0;
	float n_docs = 0;
	//getline(cin, lineInput);


	vector<string> positiveVocab;
	vector<string> negativeVocab;

	// Read in Postive Words List
	string positiveText;
	ifstream PositiveFile("positive-words.txt");
	if (PositiveFile.fail()) {
		cout << "positive-words.txt does not exist" << endl;
	}
	else {
		//cout << "File Found" << endl;
		if (PositiveFile.is_open()) {
			//cout << "file is open" << endl;
			while (getline(PositiveFile, positiveText)) {
				positiveVocab.push_back(positiveText);
			}
		}

	}
	PositiveFile.close();

	// Read in Negative Words List
	string negativeText;
	ifstream NegativeFile("negative-words.txt");
	if (NegativeFile.fail()) {
		cout << "negative-words.txt does not exist" << endl;
	}
	else {
		//cout << "File Found" << endl;
		if (NegativeFile.is_open()) {
			//cout << "file is open" << endl;
			while (getline(NegativeFile, negativeText)) {
				negativeVocab.push_back(negativeText);
			}
		}
	}
	NegativeFile.close();



	// CODE FOR TESTING
	string myText;
	while (getline(cin, lineInput)) {
		auto start = chrono::high_resolution_clock::now();
		map <string, float> countAll;
		map <string, map<string, float>> com;
		n_docs += 1;
		// Terms need to be recreated every time a tweet is read
		vector<string> terms;
		// Put input into an array
		while ((pos = lineInput.find(delimiter)) != std::string::npos) {

			// vector of terms

			token = lineInput.substr(0, pos);
			terms.push_back(token);

			//cout << token << endl;
			lineInput.erase(0, pos + delimiter.length());

			// Push into count all and count
			if (countAll.find(token) == countAll.end()) {

				countAll[token] = 1;
			}
			else {
				countAll[token] += 1;
			};
		}
		//// last one in
		//terms.push_back(lineInput);
		//// Push last one into count all and count
		//if (countAll.find(lineInput) == countAll.end()) {

		//	countAll[lineInput] = 1;
		//}
		//else {
		//	countAll[lineInput] += 1;
		//};

		// Set Ndocs
		n_docs = stoi(lineInput);

		// Sort Common Terms
		std::sort(terms.begin(), terms.end());

		// Build co - occurrence matrix 
		for (unsigned int i = 0; i < terms.size(); i++) {
			for (unsigned int j = 0; j < terms.size(); j++) {
				string w1, w2;
				w1 = terms[i];
				w2 = terms[j];
				if (w1 != w2) {

					map <string, float> w2Temp;

					all_com[w1][w2] += 1;
					com[w1][w2] = all_com[w1][w2];
				}
			}
		}



		// then make a com vector
		for (auto term1 : countAll) {
			p_t[term1.first] = term1.second / n_docs;
			for (auto term2 : com[term1.first]) {
				p_t_com[term1.first][term2.first] = com[term1.first][term2.first] / n_docs;
			}
		}

		// first make a demon vector

		int arraySize = countAll.size() * (countAll.size() - 1);

		vector<float> temp;

		float* denomVec = new float[arraySize];

		float justP_t[100] = { 0 };
		memset(justP_t, 0, 4 * countAll.size());

		
		// Make this paralell
		count = 0;
		for (auto term : p_t) {
			justP_t[count] = term.second;
			count++;
		}
	

	/*	float* justP_t = &temp[0];*/

		memset(denomVec, 0.1, 4 * arraySize);



		// Add vectors in parallel.
		hipError_t cudaStatus = denomCuda(justP_t, denomVec, p_t.size(), arraySize);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "kernal failed!");
			return 1;
		}


	//cout << denomVec[0] << endl;
	//cout << denomVec[1] << endl;
	//cout << denomVec[2] << endl;
	//cout << denomVec[3] << endl;
	//cout << denomVec[4] << endl;
	//cout << justP_t[5] << endl;
	//cout << justP_t[6] << endl;
	//cout << justP_t[7] << endl;
	//cout << justP_t[8] << endl;
	//cout << justP_t[9] << endl;
	//cout << justP_t[10] << endl;
	//cout << justP_t[11] << endl;
	//cout << justP_t[12] << endl;
		delete[] justP_t;
		delete[] denomVec;

		for (auto term1 : p_t) {
			for (auto term2 : com[term1.first]) {
				try {
					float denom;
					denom = p_t[term1.first] * p_t[term2.first];
					pmi[term1.first][term2.first] = log2(p_t_com[term1.first][term2.first] / denom);
				}
				catch (...) {
					//do nothing
				}
			}
		}


		std::map<string, float>::iterator it;
		////Remove positive and negative terms from p_t
		//for (auto term : positiveVocab) {

		//	countAll.erase(term);

		//}
		//for (auto term : negativeVocab) {
		//	countAll.erase(term);

		//}

		double tweetScore = 0;
		for (auto term : countAll) {
			float positiveAssoc = 0;
			float negativeAssoc = 0;
			bool skip = false;

			//Positive
			for (vector<string>::iterator it = positiveVocab.begin(); it != positiveVocab.end(); ++it) {




				if (pmi[term.first].count(*it) != 0) {
					positiveAssoc += pmi[term.first][*it];
				}




			}
			//Negative
			for (vector<string>::iterator it = negativeVocab.begin(); it != negativeVocab.end(); ++it) {

				if (pmi[term.first].count(*it) != 0) {
					negativeAssoc += pmi[term.first][*it];
				}



			}


			/*		auto temp = find_if(semanticOrientation.begin(), semanticOrientation.end(), [&term](std::pair<string, float> const& ref) {
						return ref.first == term.first;
						});*/


			bool duplicate = false;


			for (int i = 0; i < semanticOrientation.size(); ++i) {
				if (semanticOrientation.at(i).first == term.first) {
					semanticOrientation.at(i).second = semanticOrientation.at(i).second + positiveAssoc - negativeAssoc;
					tweetScore = tweetScore + positiveAssoc - negativeAssoc;
					duplicate = true;
				}
			}

			if (duplicate == false) {
				semanticOrientation.push_back(make_pair(term.first, (positiveAssoc - negativeAssoc)));
				tweetScore = tweetScore + positiveAssoc - negativeAssoc;
			}





		}



		//// If multiple consididate and update value
		//for (auto i = semanticOrientation.begin();i < semanticOrientation.end(); i++) {
		//	for (auto j = semanticOrientation.begin();j < semanticOrientation.end(); i++) {

		//		std::cout << i->first << endl;

		//	}
		//}


		// sort map
		std::sort(semanticOrientation.begin(), semanticOrientation.end(), cmp);

		std::cout << "Top Positive" << endl;


		for (auto i = semanticOrientation.rbegin(); i < semanticOrientation.rbegin() + 3; i++) {
			std::cout << i->first
				<< " = "
				<< i->second
				<< endl;
		}

		//print in reverse order from worst to best.
		std::cout << "Top Negative" << endl;
		for (auto i = semanticOrientation.begin();i < semanticOrientation.begin() + 3; i++) {
			std::cout << i->first
				<< " = "
				<< i->second
				<< endl;
		}
		std::cout << "Tweet Score: " << tweetScore << endl;
		auto stop = chrono::high_resolution_clock::now();
		auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
		std::cout << "Time Elapsed: " << duration.count() << " Microseconds" << endl;
		std::cout << endl;





		if (lineInput.empty()) {

			std::cout << "EOF";
			std::cout.flush();
		}
	}


	return 0;
}




// Old version might use parts of it later.
//// Add vectors in parallel.
	//hipError_t cudaStatus = sentimentAnalysis(positiveVocab, negativeVocab, allTerms, positiveVocab.size(), negativeVocab.size(), allTerms.size());
	//if (cudaStatus != hipSuccess) {
	//	fprintf(stderr, "sentimentAnalysis failed!");
	//	return 1;
	//}


		//// hipDeviceReset must be called before exiting in order for profiling and
		//// tracing tools such as Nsight and Visual Profiler to show complete traces.
		//cudaStatus = hipDeviceReset();
		//if (cudaStatus != hipSuccess) {
		//	fprintf(stderr, "hipDeviceReset failed!");
		//	return 1;
		//}




//
//
//
//	string h_positiveVocab[] = { "good", "nice", "great", "awesome", "outstanding","fantastic", "terrific", ":)", ":-)", "like", "love", "triumph", "triumphal", "triumphant", "victory" };
//	string h_negativeVocab[] = { "bad", "terrible", "crap", "useless", "hate", ":(", ":-(", "died", "dead", "defeat", "sick", "despair", "death", "deaths" };
//
//	int posCount = sizeof(h_positiveVocab), negCount = sizeof(h_negativeVocab);
//	string* d_positiveVocab, * d_negativeVocab, * d_dataKeyArray, * d_item;
//	float* d_dataValueArray;
//	const int posSize = posCount * sizeof(string);
//	const int negSize = negCount * sizeof(string);
//	hipMalloc(&d_positiveVocab, posSize);
//	hipMalloc(&d_negativeVocab, negSize);
//	hipMemcpy(d_positiveVocab, h_positiveVocab, posSize, hipMemcpyKind::hipMemcpyHostToDevice);
//	hipMemcpy(d_negativeVocab, h_negativeVocab, negSize, hipMemcpyKind::hipMemcpyHostToDevice);
//	map <string, float> out;
//	map <string, float> semanticOrientation;
//	// semantic orientation
//	for (auto item : out) {
//		float positiveAssoc = 0;
//		float negativeAssoc = 0;
//		// Split map to copy to device
//		const int dataCount = sizeof(pmi[item.first]);
//		string h_dataKeyArray[dataCount], h_item;
//		float h_dataValueArray[dataCount];
//		int tempCount = 0;
//		for (auto data : pmi[item.first]) {
//			h_dataKeyArray[tempCount] = item.first;
//			h_dataValueArray[tempCount] = item.second;
//			h_item = item.first;
//			tempCount++;
//			//Make Copy to Device
//			const int dataKeyArraySize = dataCount * sizeof(string), dataValueArraySize = dataCount * sizeof(float), itemSize = sizeof(string);
//			hipMalloc(&d_item, itemSize);
//			hipMalloc(&d_dataKeyArray, dataKeyArraySize);
//			hipMalloc(&d_dataValueArray, dataValueArraySize);
//			hipMemcpy(d_dataKeyArray, h_dataKeyArray, dataKeyArraySize, hipMemcpyKind::hipMemcpyHostToDevice);
//			hipMemcpy(d_dataValueArray, h_dataValueArray, dataValueArraySize, hipMemcpyKind::hipMemcpyHostToDevice);
//			sentimentAnalysis <<<1, 10 >>> (d_positiveVocab, d_negativeVocab, d_dataKeyArray, d_dataValueArray, d_item);
//		}
//
//
//		semanticOrientation[item.first] = positiveAssoc - negativeAssoc;
//
//	}
//
//
//
//	// sort map
//	//sort(semanticOrientation.begin(), semanticOrientation.end(), [](const auto& x, const auto& y) {return x.second < y.second;});
//	sort(semanticOrientation);
//
//
//	cout << "Top Positive";
//	for (auto i = semanticOrientation.begin();i != semanticOrientation.end(); i++) {
//		cout << i->first
//			<< " = "
//			<< i->second
//			<< endl;
//	}
//
//	cout << endl;
//
//	//print in reverse order from worst to best.
//	cout << "Top Negative";
//	for (auto i = semanticOrientation.rbegin(); i != semanticOrientation.rend(); i++) {
//		cout << i->first
//			<< " = "
//			<< i->second
//			<< endl;
//	}
//
//	cout.flush();
//	std::system("pause");
//	return;
//}

// Sort terms
//sort(terms.begin(), terms.end());